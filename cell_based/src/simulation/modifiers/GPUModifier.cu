#include "hip/hip_runtime.h"
/*

Copyright (c) 2005-2024, University of Oxford.
All rights reserved.

University of Oxford means the Chancellor, Masters and Scholars of the
University of Oxford, having an administrative office at Wellington
Square, Oxford OX1 2JD, UK.

This file is part of Chaste.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:
 * Redistributions of source code must retain the above copyright notice,
   this list of conditions and the following disclaimer.
 * Redistributions in binary form must reproduce the above copyright notice,
   this list of conditions and the following disclaimer in the documentation
   and/or other materials provided with the distribution.
 * Neither the name of the University of Oxford nor the names of its
   contributors may be used to endorse or promote products derived from this
   software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE
GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION)
HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT
OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

*/

#include "GPUModifier.cuh"
#include "MeshBasedCellPopulation.hpp"

FLAMEGPU_AGENT_FUNCTION(output_location, flamegpu::MessageNone, flamegpu::MessageSpatial2D) {
    FLAMEGPU->message_out.setVariable<float>("x", FLAMEGPU->getVariable<float>("x"));
    FLAMEGPU->message_out.setVariable<float>("y", FLAMEGPU->getVariable<float>("y"));
    FLAMEGPU->message_out.setVariable<float>("radius", FLAMEGPU->getVariable<float>("radius"));
    return flamegpu::ALIVE;
}

// Models repulsion force without division/apoptosis
FLAMEGPU_AGENT_FUNCTION(compute_force_meineke_spring, flamegpu::MessageSpatial2D, flamegpu::MessageNone) {
    const double x = FLAMEGPU->getVariable<float>("x");
    const double y = FLAMEGPU->getVariable<float>("y");
    float x_force = 0.0;
    float y_force = 0.0;
    float radius = FLAMEGPU->getVariable<float>("radius");

    for (const auto& message : FLAMEGPU->message_in(x, y)) {
        float other_x = message.getVariable<float>("x");
        float other_y = message.getVariable<float>("y");
        float other_radius = message.getVariable<float>("radius");
        
        // Compute unit distance
        float x_dist = other_x - x;
        float y_dist = other_y - y;
        float distance_between_nodes = sqrt(x_dist * x_dist + y_dist * y_dist);

        float unit_x = x_dist / distance_between_nodes;
        float unit_y = y_dist / distance_between_nodes;
        
        // Only compute force if within cutoff distance and for positive distance
        const float cutoff_length = 1.5f;
        if (distance_between_nodes < cutoff_length && distance_between_nodes > 0.0f) {

            // Compute rest length
            const float rest_length = radius + other_radius; 
            const float rest_length_final = rest_length;
            
            // TODO: Should check here if newly divided or apoptosis happening


            // Compute the force
            float overlap = distance_between_nodes - rest_length;
            bool is_closer_than_rest_length = (overlap <= 0);
            const float spring_stiffness = 15.0f;
            const float multiplication_factor = 1.0f;

            
            // A reasonably stable simple force law
            if (is_closer_than_rest_length) //overlap is negative
            {
                //assert(overlap > -rest_length_final);
                x_force += multiplication_factor * spring_stiffness * unit_x * rest_length_final* log(1.0 + overlap/rest_length_final);
                y_force  = multiplication_factor * spring_stiffness * unit_y * rest_length_final* log(1.0 + overlap/rest_length_final);
            }
            else
            {
                double alpha = 5.0;
                x_force += multiplication_factor * spring_stiffness * unit_x * overlap * exp(-alpha * overlap/rest_length_final);
                y_force += multiplication_factor * spring_stiffness * unit_y * overlap * exp(-alpha * overlap/rest_length_final);
            }
        }

        
    }
    FLAMEGPU->setVariable<float>("x_force", x_force);        
    FLAMEGPU->setVariable<float>("y_force", y_force);        
    return flamegpu::ALIVE;
}

template<unsigned DIM>
GPUModifier<DIM>::GPUModifier()
    : AbstractCellBasedSimulationModifier<DIM>(),
    mpFlameGPUModel(nullptr),
    mpCellAgentDescription(nullptr),
    mpFlameGPUSimulation(nullptr)
{
}

template<unsigned DIM>
GPUModifier<DIM>::~GPUModifier()
{
}

template<unsigned DIM>
void GPUModifier<DIM>::UpdateAtEndOfTimeStep(AbstractCellPopulation<DIM,DIM>& rCellPopulation)
{
    // Reset the simulation
    mpFlameGPUSimulation->resetStepCounter();

    // Extract cell locations from chaste
    // Get number of cells & resize agent vector to match
    unsigned int numCells = rCellPopulation.rGetMesh().GetNumNodes();
    mpCellAgentVector->resize(numCells);

    // Set the positions and clear the forces
    auto& rMesh = rCellPopulation.rGetMesh();
    auto& cellVector = *mpCellAgentVector; // Grab ref to vector for easier indexing
    unsigned int i = 0;
    for (auto iter = rMesh.GetNodeIteratorBegin(); iter != rMesh.GetNodeIteratorEnd(); ++iter) {
      cellVector[i].setVariable<float>("x", iter->rGetLocation()[0]);
      cellVector[i].setVariable<float>("y", iter->rGetLocation()[1]);
      cellVector[i].setVariable<float>("radius", 1.5f);
      cellVector[i].setVariable<float>("x_force", 0.0f);
      cellVector[i].setVariable<float>("y_force", 0.0f);
      i++;
    }

    // Create cell population for FlameGPU simulation
    mpFlameGPUSimulation->setPopulationData(*mpCellAgentVector);

    // Run the simulation
    mpFlameGPUSimulation->simulate();

    // Extract results
    flamegpu::AgentVector out_pop(*mpCellAgentDescription);
    mpFlameGPUSimulation->getPopulationData(*mpCellAgentVector);

    // Apply results to chaste - TODO: Assumes no change in pop size. Should always be true for force resolution?
    i = 0;
    for (auto iter = rMesh.GetNodeIteratorBegin(); iter != rMesh.GetNodeIteratorEnd(); ++iter) {
        iter->rGetModifiableLocation()[0] = cellVector[i].getVariable<float>("x");
        iter->rGetModifiableLocation()[1] = cellVector[i].getVariable<float>("y");
        i++;
    }
}

template<unsigned DIM>
void GPUModifier<DIM>::SetupSolve(AbstractCellPopulation<DIM,DIM>& rCellPopulation, std::string outputDirectory)
{
    mpFlameGPUModel = std::make_unique<flamegpu::ModelDescription>("ForceResolutionModel");
    
    // Define an agent
    mpCellAgentDescription = std::make_unique<flamegpu::AgentDescription>(mpFlameGPUModel->newAgent("cell"));
    mpCellAgentDescription->newVariable<float>("x");
    mpCellAgentDescription->newVariable<float>("y");
    mpCellAgentDescription->newVariable<float>("radius");
    mpCellAgentDescription->newVariable<float>("x_force");
    mpCellAgentDescription->newVariable<float>("y_force");
    
    // Define the location message
    flamegpu::MessageSpatial2D::Description location_message = mpFlameGPUModel->newMessage<flamegpu::MessageSpatial2D>("location_message");
    //location_message.newVariable<float>("x"); // Implicit for spatial message
    //location_message.newVariable<float>("y"); // Implicit for spatial message
    location_message.newVariable<float>("radius");
    location_message.setMin(-500.0, -500.0);
    location_message.setMax(500.0, 500.0);
    location_message.setRadius(1.5);

    // Agent functions
    flamegpu::AgentFunctionDescription output_location_desc = mpCellAgentDescription->newFunction("output_location", output_location);
    output_location_desc.setMessageOutput("location_message");
    
    flamegpu::AgentFunctionDescription compute_force_desc = mpCellAgentDescription->newFunction("csfompute_force_meineke_spring", compute_force_meineke_spring);
    compute_force_desc.setMessageInput("location_message");

    compute_force_desc.dependsOn(output_location_desc);
    
    // Set execution root
    mpFlameGPUModel->addExecutionRoot(output_location_desc);
    
    // Generate execution plan
    mpFlameGPUModel->generateLayers();
      
    // Construct a simulation object from the model and configure it to run for a single step
    mpFlameGPUSimulation = std::make_unique<flamegpu::CUDASimulation>(*mpFlameGPUModel);
    mpFlameGPUSimulation->SimulationConfig().steps = 1;
    
    // Allocate a vector for transferring agent data between host & device
    mpCellAgentVector = std::make_unique<flamegpu::AgentVector>(*mpCellAgentDescription);
}


template<unsigned DIM>
void GPUModifier<DIM>::OutputSimulationModifierParameters(out_stream& rParamsFile)
{
    // No parameters to output, so just call method on direct parent class
    AbstractCellBasedSimulationModifier<DIM>::OutputSimulationModifierParameters(rParamsFile);
}

// Explicit instantiation
template class GPUModifier<1>;
template class GPUModifier<2>;
template class GPUModifier<3>;

// Serialization for Boost >= 1.36
#include "SerializationExportWrapperForCpp.hpp"
EXPORT_TEMPLATE_CLASS_SAME_DIMS(GPUModifier)

